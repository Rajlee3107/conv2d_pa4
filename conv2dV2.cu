#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iomanip>

#define BLOCK_SIZE 16

void readInput(const char* fileName, float*& h_input, int& H, int& W, int& N) {
    std::ifstream inFile(fileName);
    if (!inFile) {
        std::cerr << "Error: Could not open input file " << fileName << std::endl;
        exit(EXIT_FAILURE);
    }

    inFile >> H >> W >> N;
    h_input = (float*)malloc(N * H * W * sizeof(float));

    for (int n = 0; n < N; ++n)
        for (int i = 0; i < H; ++i)
            for (int j = 0; j < W; ++j)
                inFile >> h_input[n * H * W + i * W + j];
}

void readFilter(const char* fileName, float*& h_filter, int& K, int& R) {
    std::ifstream inFile(fileName);
    if (!inFile) {
        std::cerr << "Error: Could not open filter file " << fileName << std::endl;
        exit(EXIT_FAILURE);
    }

    inFile >> R;
    inFile >> K;
    h_filter = (float*)malloc(K * R * R * sizeof(float));

    for (int k = 0; k < K; ++k)
        for (int i = 0; i < R; ++i)
            for (int j = 0; j < R; ++j)
                inFile >> h_filter[k * R * R + i * R + j];
}

void writeOutput(float* h_output, int H, int W, int N, int K) {
    for (int nk = 0; nk < N * K; ++nk) {
        for (int i = 0; i < H; ++i) {
            for (int j = 0; j < W; ++j) {
                std::cout << std::fixed << std::setprecision(3) << h_output[nk * H * W + i * W + j];
                if (j != W - 1) std::cout << " ";
            }
            std::cout << std::endl;
        }
    }
}

__global__ void conv2d_shared_kernel(float* input, float* filter, float* output, int H, int W, int N, int K, int R) {
    extern __shared__ float sh_input[];

    int tx = threadIdx.x, ty = threadIdx.y;
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;
    int nk = blockIdx.z;
    int n = nk / K;
    int k = nk % K;
    int pad = R / 2;

    int sh_W = BLOCK_SIZE + 2 * pad;
    int local_r = ty + pad;
    int local_c = tx + pad;

    float* sh = sh_input;

    // Initialize all shared memory tile to 0
    for (int i = ty; i < sh_W; i += blockDim.y)
        for (int j = tx; j < sh_W; j += blockDim.x)
            sh[i * sh_W + j] = 0.0f;

    __syncthreads();

    // Load center and halos into shared memory with bounds checks
    if ((row < H) && (col < W))
        sh[local_r * sh_W + local_c] = input[n * H * W + row * W + col];

    if (tx < pad && col >= pad)
        sh[local_r * sh_W + (local_c - pad)] = input[n * H * W + row * W + (col - pad)];
    if (tx >= BLOCK_SIZE - pad && col + pad < W)
        sh[local_r * sh_W + (local_c + pad)] = input[n * H * W + row * W + (col + pad)];
    if (ty < pad && row >= pad)
        sh[(local_r - pad) * sh_W + local_c] = input[n * H * W + (row - pad) * W + col];
    if (ty >= BLOCK_SIZE - pad && row + pad < H)
        sh[(local_r + pad) * sh_W + local_c] = input[n * H * W + (row + pad) * W + col];

    if (tx < pad && ty < pad && row >= pad && col >= pad)
        sh[(local_r - pad) * sh_W + (local_c - pad)] = input[n * H * W + (row - pad) * W + (col - pad)];
    if (tx >= BLOCK_SIZE - pad && ty < pad && row >= pad && col + pad < W)
        sh[(local_r - pad) * sh_W + (local_c + pad)] = input[n * H * W + (row - pad) * W + (col + pad)];
    if (tx < pad && ty >= BLOCK_SIZE - pad && row + pad < H && col >= pad)
        sh[(local_r + pad) * sh_W + (local_c - pad)] = input[n * H * W + (row + pad) * W + (col - pad)];
    if (tx >= BLOCK_SIZE - pad && ty >= BLOCK_SIZE - pad && row + pad < H && col + pad < W)
        sh[(local_r + pad) * sh_W + (local_c + pad)] = input[n * H * W + (row + pad) * W + (col + pad)];

    __syncthreads();

    if (row < H && col < W) {
        float sum = 0.0f;
        for (int i = 0; i < R; ++i)
            for (int j = 0; j < R; ++j)
                sum += sh[(local_r + i - pad) * sh_W + (local_c + j - pad)] * filter[k * R * R + i * R + j];
        output[nk * H * W + row * W + col] = sum;
    }
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        std::cerr << "ERROR: Number of arguments < 3" << std::endl;
        return -1;
    }

    int H, W, N, K, R;
    float *h_input, *h_filter, *h_output;
    readInput(argv[1], h_input, H, W, N);
    readFilter(argv[2], h_filter, K, R);

    float *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, N * H * W * sizeof(float));
    hipMalloc(&d_filter, K * R * R * sizeof(float));
    hipMalloc(&d_output, N * K * H * W * sizeof(float));

    hipMemcpy(d_input, h_input, N * H * W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, K * R * R * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((W + BLOCK_SIZE - 1) / BLOCK_SIZE, (H + BLOCK_SIZE - 1) / BLOCK_SIZE, N * K);
    int sharedMemSize = (BLOCK_SIZE + 2 * (R / 2)) * (BLOCK_SIZE + 2 * (R / 2)) * sizeof(float);

    conv2d_shared_kernel<<<gridDim, blockDim, sharedMemSize>>>(d_input, d_filter, d_output, H, W, N, K, R);
    hipDeviceSynchronize();

    h_output = (float*)malloc(N * K * H * W * sizeof(float));
    hipMemcpy(h_output, d_output, N * K * H * W * sizeof(float), hipMemcpyDeviceToHost);

    writeOutput(h_output, H, W, N, K);

    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    free(h_input);
    free(h_filter);
    free(h_output);

    return 0;
}

